#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			100000	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100	// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif


float chooseRand( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}


__global__  void ArrayMul( float *A, float *B, float *C, float *D )
{
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	
	// randomize the location and radius of the circle:
	float xc = A[gid];
	float yc = B[gid];
	float  r = C[gid];

	// solve for the intersection using the quadratic formula:
	float a = 2.;
	float b = -2.*( xc + yc );
	float c = xc*xc + yc*yc - r*r;
	float d = b*b - 4.*a*c;
	//If d is less than 0., then the circle was completely missed. (Case A) Continue on to the next trial in the for-loop.

	if (d >= 0) {
		// hits the circle:
		// get the first intersection:
		d = sqrt( d );
		float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
		float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
		float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection
	

		//If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Continue on to the next trial in the for-loop.
		if(tmin >= 0) {

			// where does it intersect the circle?
			float xcir = tmin;
			float ycir = tmin;

			// get the unitized normal vector at the point of intersection:
			float nx = xcir - xc;
			float ny = ycir - yc;
			float n = sqrt( nx*nx + ny*ny );
			nx /= n;	// unit vector
			ny /= n;	// unit vector

			// get the unitized incoming vector:
			float inx = xcir - 0.;
			float iny = ycir - 0.;
			float in = sqrt( inx*inx + iny*iny );
			inx /= in;	// unit vector
			iny /= in;	// unit vector

			// get the outgoing (bounced) vector:
			float dot = inx*nx + iny*ny;
			float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
			float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

			// find out if it hits the infinite plate:
			float t = ( 0. - ycir ) / outy;

			//If t is less than 0., then the reflected beam went up instead of down. Continue on to the next trial in the for-loop.
			//Otherwise, this beam hit the infinite plate. (Case D) Increment the number of hits and continue on to the next trial in the for-loop.
			if (t >= 0) {
				D[gid] = 1;
			}
			else {
				D[gid] = 0;
			}
		}
	}
}


// main program:

int
main( int argc, char* argv[ ] )
{
	//int dev = findCudaDevice(argc, (const char **)argv);


	const float XCMIN =	  0.;
	const float XCMAX =	 2.0;
	const float YCMIN =	  0.;
	const float YCMAX =	 2.0;
	const float RMIN  =	 0.5;
	const float RMAX  =	 2.0;

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE ];
	float * hD = new float [ SIZE ];

	for( int n = 0; n < SIZE; n++ )
     {       
     	hA[n] = chooseRand( XCMIN, XCMAX );
        hB[n] = chooseRand( YCMIN, YCMAX );
     	hC[n] = chooseRand(  RMIN,  RMAX ); 
     	hD[n] = 0.;
     }

	// allocate device memory:

	float *dA, *dB, *dC, *dD;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE, 1, 1 );
	dim3 dimsD( SIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dD), SIZE*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dC, hC, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dD, hD, SIZE*sizeof(float), hipMemcpyHostToDevice );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        ArrayMul<<< grid, threads >>>( dA, dB, dC, dD );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	//fprintf( stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, SIZE*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	status = hipMemcpy( hD, dD, SIZE*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );

	double hits = 0;
	for ( int i = 0; i < SIZE; i++) {
		hits += hD[i];
	}	

	//fprintf(stderr, "Size: %d\n", SIZE);
	//double probability = hits/SIZE;
	fprintf(stderr, "Probability:%8.2lf\n", hits/SIZE);
	//printf("%d\t%d\t%10.2lf\t%8.2lf\n", BLOCKSIZE, SIZE, megaMultsPerSecond, hits/SIZE);
	printf("%10.2lf\t", megaMultsPerSecond);

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;
	delete [ ] hD;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );
	status = hipFree( dD );
		checkCudaErrors( status );
	

	return 0;
}

